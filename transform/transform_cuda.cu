#include "hip/hip_runtime.h"
extern "C" {
  #include "transform_cuda.h"
}

/*********************************
*         INVERT_COLORS          *
*********************************/
void invert_colors(unsigned char *pixmap, int width, int height, int depth)
{
  unsigned char *beg = pixmap;
  if(depth == 24) {
    for(int i = 0; i < (height); ++i) {
      for(int n = 0; n < (width); ++n) {
        for(int x = 0; x < 3; ++x) {
          *pixmap = 255 - (*pixmap);
          ++pixmap;
        }
        ++pixmap;
      }
    }
  }
  pixmap = beg;
}

/*********************************
*         GAMMA                  *
*********************************/

__global__
void gamma_subset(void *args, void *d_pixmap, void *d_pixmapmod)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  struct Gamargs *pic = (struct Gamargs *)args;
  unsigned char *pixmap = (unsigned char *)d_pixmap;
  unsigned char *pixmapmod = (unsigned char *)d_pixmapmod;
  unsigned char tmp;

  if(pic->depth == 24) {
    tmp = pow((float)(*(pixmap + i)) / 255, (float)(1 / pic->gam)) * 255;
    if((tmp > *(pixmapmod + i)) && (pic->gam < pic->prevgam) && (pic->gam < 1.0)) {
      tmp = 0;
    }
    if((tmp < *(pixmapmod + i)) && (pic->gam > pic->prevgam) && (pic->gam > 1.0)) {
      tmp = 255;
    }
    *(pixmapmod + i) = tmp;
  }
}

extern "C"
  void *apply_gamma(void *args)
  {
    struct Gamargs *oldargs = (struct Gamargs *)args;
    struct Gamargs *newargs, *d_newargs;
    unsigned char *d_pixmap, *d_pixmapmod;
    int numpixels;

    newargs = (struct Gamargs *)malloc(sizeof(struct Gamargs));
    newargs->width = oldargs->width;
    newargs->height = oldargs->height;
    newargs->depth = oldargs->depth;
    newargs->gam = oldargs->gam;
    newargs->prevgam = oldargs->prevgam;
    newargs->pixmap = oldargs->pixmap;
    newargs->pixmapmod = oldargs->pixmapmod;
    
    if(newargs->depth == 24) {
      printf("Setting numpixels\n");
      numpixels = (newargs->width) * (newargs->height) * 4;
    }

    // Allocate the memory on the GPU and run the kernel
    hipMalloc((void **)(&d_newargs), sizeof(struct Gamargs));
    hipMalloc((void **)(&d_pixmap), sizeof(unsigned char) * 4 * newargs->height * newargs->width);
    hipMalloc((void **)(&d_pixmapmod), sizeof(unsigned char) * 4 * newargs->height * newargs->width);
    hipMemcpy(d_newargs, newargs, sizeof(struct Gamargs), hipMemcpyHostToDevice);
    hipMemcpy(d_pixmap, newargs->pixmap, sizeof(unsigned char) * 4 * newargs->height * newargs->width, hipMemcpyHostToDevice);
    hipMemcpy(d_pixmapmod, newargs->pixmapmod, sizeof(unsigned char) * 4 * newargs->height * newargs->width, hipMemcpyHostToDevice);
    printf("%d\n", numpixels);
    printf("spawning %d blocks\n", (numpixels + 255) / 256);
    gamma_subset<<<(numpixels + 255) / 256, 256>>>((void *)d_newargs, (void *)d_pixmap, (void *)d_pixmapmod);

    // Pass the data back
    hipMemcpy(newargs, d_newargs, sizeof(struct Gamargs), hipMemcpyDeviceToHost);
    hipMemcpy(newargs->pixmap, d_pixmap, sizeof(unsigned char) * 4 * newargs->height * newargs->width, hipMemcpyDeviceToHost);
    hipMemcpy(newargs->pixmapmod, d_pixmapmod, sizeof(unsigned char) * 4 * newargs->height * newargs->width, hipMemcpyDeviceToHost);

    return NULL;
  }

