#include "hip/hip_runtime.h"
extern "C" {
  #include "transform_cuda.h"
}

/*********************************
*         GAMMA                  *
*********************************/

// Function for each thread to run
void *cpu_gamma_subset(void *args)
{
  struct Cpuargs *pic = (struct Cpuargs *)args;
  unsigned char tmp;
  for(int i = 0; i < pic->size; ++i) {
    tmp = pow((float)(*(pic->pixmap)) / 255, (float)(1 / pic->gam)) * 255;
    if((tmp > *(pic->pixmapmod)) && (pic->gam < pic->prevgam) && (pic->gam < 1.0)) {
      tmp = 0;
    }
    if((tmp < *(pic->pixmapmod)) && (pic->gam > pic->prevgam) && (pic->gam > 1.0)) {
      tmp = 255;
    }
    *(pic->pixmapmod) = tmp;
    ++(pic->pixmap);
    ++(pic->pixmapmod);
  }
  return NULL;
}


//cuda kernel
__global__
void gpu_gamma_subset(void *args)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  struct Gpuargs *pic = (struct Gpuargs *)args;
  unsigned char *pixmap = (unsigned char *)pic->pixmap;
  unsigned char *pixmapmod = (unsigned char *)pic->pixmapmod;
  unsigned char tmp;

  if(i < (pic->size)) {
    tmp = pow((float)(*(pixmap + i)) / 255, (float)(1 / pic->gam)) * 255;
    if((tmp > *(pixmapmod + i)) && (pic->gam < pic->prevgam) && (pic->gam < 1.0)) {
      tmp = 0;
    }
    if((tmp < *(pixmapmod + i)) && (pic->gam > pic->prevgam) && (pic->gam > 1.0)) {
      tmp = 255;
    }
    *(pixmapmod + i) = tmp;
  }
}

extern "C"
void *apply_gamma(void *args)
{
  struct Gamargs *oldargs = (struct Gamargs *)args;
  unsigned char *pixmap = oldargs->pixmap;
  unsigned char *pixmapmod = oldargs->pixmapmod;

  // Figure how much of the image the CPU/GPU is doing
  int gpusize, cpusize;
  if(oldargs->depth == 24) {
    gpusize = oldargs->height * oldargs->width * 4 * oldargs->split;
    cpusize = (oldargs->height * oldargs->width * 4) - gpusize;
  }

  // Set up data structures for the CPU threads
  int numthreads = oldargs->numthreads;
  struct Cpuargs *cpuargs;
  pthread_t *tids = (pthread_t *)malloc(sizeof(pthread_t) * numthreads);

  // Create all of the threads
  for(int i = 0; i < numthreads; ++i) {
    cpuargs = (struct Cpuargs *)malloc(sizeof(struct Cpuargs));
    cpuargs->size = cpusize / numthreads;
    cpuargs->depth = oldargs->depth;
    cpuargs->gam = oldargs->gam;
    cpuargs->prevgam = oldargs->prevgam;
    cpuargs->pixmap = oldargs->pixmap + gpusize + (i * (cpuargs->size));
    cpuargs->pixmapmod = oldargs->pixmapmod + gpusize + (i * (cpuargs->size));
    pthread_create(tids, NULL, cpu_gamma_subset, cpuargs);
    ++tids;
  }
  for(int i = 0; i < numthreads; ++i) {
    --tids;
  }
  
  // Set up the GPU data structures
  struct Gpuargs *gpuargs, *d_gpuargs;
  gpuargs = (struct Gpuargs *)malloc(sizeof(struct Gpuargs));
  gpuargs->depth = oldargs->depth;
  gpuargs->size = gpusize;
  gpuargs->gam = oldargs->gam;
  gpuargs->prevgam = oldargs->prevgam;

  // Allocate and move data to the GPU
  hipMalloc((void **)(&d_gpuargs), sizeof(struct Gpuargs));
  hipMalloc((void **)(&gpuargs->pixmap), sizeof(unsigned char) * gpuargs->size);
  hipMalloc((void **)(&gpuargs->pixmapmod), sizeof(unsigned char) * gpuargs->size);
  hipMemcpy(d_gpuargs, gpuargs, sizeof(struct Gpuargs), hipMemcpyHostToDevice);
  hipMemcpy(gpuargs->pixmap, pixmap, sizeof(unsigned char) * gpuargs->size, hipMemcpyHostToDevice);
  hipMemcpy(gpuargs->pixmapmod, pixmapmod, sizeof(unsigned char) * gpuargs->size, hipMemcpyHostToDevice);

  // Run!
  gpu_gamma_subset<<<(gpuargs->size + (oldargs->threadsperblock -1)) / oldargs->threadsperblock, oldargs->threadsperblock>>>((void *)d_gpuargs);

  // Pass the data back
  hipMemcpy(oldargs->pixmapmod, gpuargs->pixmapmod, sizeof(unsigned char) * gpuargs->size, hipMemcpyDeviceToHost);

  // Sync all of the threads
  for(int i = 0; i < numthreads; ++i) {
    pthread_join(*tids, NULL);
    ++tids;
  }

  return NULL;
}
